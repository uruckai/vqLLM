#include "hip/hip_runtime.h"
/**
 * @file decoder_gpu.cu
 * @brief GPU decoder CUDA kernel
 */

#include "format.h"
#include <hip/hip_runtime.h>
#include <cstdio>

namespace codec {

/**
 * Simple differential decode on GPU (per-tile)
 */
__device__ void ransDecodeDevice(
    const uint8_t* stream, size_t stream_size,
    const uint32_t* freqs, int8_t* output, size_t output_size)
{
    // Debug: print first call info
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("GPU Decode: stream_size=%zu, output_size=%zu\n", stream_size, output_size);
        if (stream_size >= 4) {
            printf("  First 4 header bytes: %u %u %u %u\n", 
                   stream[0], stream[1], stream[2], stream[3]);
        }
    }
    
    // Read size header (4 bytes)
    if (stream_size < 4) {
        printf("GPU Decode ERROR: stream_size < 4\n");
        return;
    }

    uint32_t data_size = (stream[0] << 0) | (stream[1] << 8) |
                        (stream[2] << 16) | (stream[3] << 24);
    
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("  data_size from header: %u\n", data_size);
    }

    // Read differential data
    size_t read_size = min(data_size, static_cast<uint32_t>(output_size));
    int32_t prev = 0;  // Use int32 to avoid overflow
    
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("  read_size: %zu, first 4 data bytes: %u %u %u %u\n", 
               read_size, stream[4], stream[5], stream[6], stream[7]);
    }

    for (size_t i = 0; i < read_size && (i + 4) < stream_size; i++) {
        uint8_t diff_byte = stream[i + 4];
        // Convert from uint8 centered at 128 to signed diff
        // Encoder does: (diff + 128) & 0xFF, which wraps negative diffs
        // To decode: treat (diff_byte - 128) as an 8-bit signed value
        int32_t diff_temp = static_cast<int32_t>(diff_byte) - 128;
        // Convert to proper signed value: if > 127, it's actually negative
        int32_t diff = (diff_temp > 127) ? (diff_temp - 256) : diff_temp;
        int32_t current = prev + diff;
        output[i] = static_cast<int8_t>(current);
        prev = current;
        
        // Debug first few values
        if (threadIdx.x == 0 && blockIdx.x == 0 && i < 4) {
            printf("    [%zu] byte=%u, diff_temp=%d, diff=%d, prev=%d, current=%d, out=%d\n",
                   i, diff_byte, diff_temp, diff, static_cast<int>(prev - diff), current, output[i]);
        }
    }
    
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("  Decoded %zu values\n", read_size);
    }
}

/**
 * Reconstruct tile from residual using predictor
 */
__device__ void reconstructDevice(
    int8_t* tile, uint32_t rows, uint32_t cols,
    const int8_t* left, const int8_t* top, PredictorMode mode)
{
    for (uint32_t r = 0; r < rows; r++) {
        for (uint32_t c = 0; c < cols; c++) {
            int idx = r * cols + c;
            int8_t pred = 0;
            
            switch (mode) {
                case PRED_LEFT:
                    pred = (c > 0) ? tile[idx - 1] : (left ? left[r] : 0);
                    break;
                    
                case PRED_TOP:
                    pred = (r > 0) ? tile[idx - cols] : (top ? top[c] : 0);
                    break;
                    
                case PRED_AVG: {
                    int8_t left_val = (c > 0) ? tile[idx - 1] : (left ? left[r] : 0);
                    int8_t top_val = (r > 0) ? tile[idx - cols] : (top ? top[c] : 0);
                    pred = (left_val + top_val) / 2;
                    break;
                }
                
                case PRED_PLANAR: {
                    int8_t left_val = (c > 0) ? tile[idx - 1] : (left ? left[r] : 0);
                    int8_t top_val = (r > 0) ? tile[idx - cols] : (top ? top[c] : 0);
                    int8_t tl_val = 0;
                    if (r > 0 && c > 0) {
                        tl_val = tile[idx - cols - 1];
                    }
                    pred = left_val + top_val - tl_val;
                    break;
                }
            }
            
            // Residual -> Original
            tile[idx] = tile[idx] + pred;
        }
    }
}

/**
 * Main decode kernel - one thread block per tile
 */
__global__ void decodeKernel(
    const uint8_t* compressed,
    const Header* header,
    const TileMetadata* tile_metadata,
    int8_t* output)
{
    // Each block handles one tile
    uint32_t tile_idx = blockIdx.x;
    uint32_t num_tiles = header->num_tiles_row * header->num_tiles_col;
    
    if (tile_idx >= num_tiles) return;
    
    // Calculate tile position
    uint32_t ty = tile_idx / header->num_tiles_col;
    uint32_t tx = tile_idx % header->num_tiles_col;
    
    uint32_t row_start = ty * header->tile_size;
    uint32_t col_start = tx * header->tile_size;
    uint32_t tile_rows = min(header->tile_size, header->output_rows - row_start);
    uint32_t tile_cols = min(header->tile_size, header->output_cols - col_start);
    
    // Shared memory for tile
    extern __shared__ int8_t tile_data[];
    
    // Thread 0 does rANS decode
    if (threadIdx.x == 0) {
        const TileMetadata& meta = tile_metadata[tile_idx];
        
        if (tile_idx == 0) {
            printf("Tile 0 metadata: data_offset=%u, data_size=%u, predictor=%d\n",
                   meta.data_offset, meta.data_size, meta.predictor_mode);
        }
        
        const uint8_t* tile_stream = compressed + meta.data_offset;
        
        ransDecodeDevice(tile_stream, meta.data_size, meta.freq_table, 
                        tile_data, tile_rows * tile_cols);
    }
    __syncthreads();
    
    // Thread 0 does reconstruction
    if (threadIdx.x == 0) {
        const TileMetadata& meta = tile_metadata[tile_idx];
        
        // Context pointers (simplified - no boundary handling for now)
        const int8_t* left = nullptr;
        const int8_t* top = nullptr;
        
        reconstructDevice(tile_data, tile_rows, tile_cols, left, top,
                         static_cast<PredictorMode>(meta.predictor_mode));
    }
    __syncthreads();
    
    // All threads cooperate to write output
    for (uint32_t i = threadIdx.x; i < tile_rows * tile_cols; i += blockDim.x) {
        uint32_t r = i / tile_cols;
        uint32_t c = i % tile_cols;
        output[(row_start + r) * header->output_cols + col_start + c] = tile_data[i];
    }
}

} // namespace codec

/**
 * Host launch function
 */
extern "C" void launchDecodeKernel(
    const uint8_t* d_compressed,
    const codec::Header* d_header,
    const codec::TileMetadata* d_metadata,
    int8_t* d_output,
    hipStream_t stream)
{
    // Copy header to host to get dimensions
    codec::Header h_header;
    hipMemcpy(&h_header, d_header, sizeof(codec::Header), hipMemcpyDeviceToHost);
    
    uint32_t num_tiles = h_header.num_tiles_row * h_header.num_tiles_col;
    size_t shared_mem = h_header.tile_size * h_header.tile_size * sizeof(int8_t);
    
    // Launch one block per tile, 256 threads per block
    codec::decodeKernel<<<num_tiles, 256, shared_mem, stream>>>(
        d_compressed, d_header, d_metadata, d_output
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

