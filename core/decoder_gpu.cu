#include "hip/hip_runtime.h"
/**
 * @file decoder_gpu.cu
 * @brief GPU decoder CUDA kernel
 */

#include "format.h"
#include <hip/hip_runtime.h>
#include <cstdio>

namespace codec {

/**
 * rANS symbol structure (matches host)
 */
struct RANSSymbol {
    uint16_t start;
    uint16_t freq;
};

/**
 * Simple differential decode (no rANS for now - it expands small data)
 */
__device__ void ransDecodeDevice(
    const uint8_t* stream, size_t stream_size,
    const RANSSymbol* symbols, int8_t* output, size_t output_size)
{
    // Stream format: 4 bytes (size header) + differential data
    if (stream_size < 4) return;

    // Read output size
    uint32_t stored_size = (stream[0] << 0) | (stream[1] << 8) |
                          (stream[2] << 16) | (stream[3] << 24);
    if (stored_size != output_size) return;

    // Read differential data directly (no rANS decoding)
    size_t read_size = min(stored_size, static_cast<uint32_t>(output_size));
    
    // Apply differential decoding
    int32_t prev = 0;
    for (size_t i = 0; i < read_size && (i + 4) < stream_size; i++) {
        uint8_t diff_byte = stream[i + 4];
        int32_t diff_temp = static_cast<int32_t>(diff_byte) - 128;
        int32_t diff = (diff_temp > 127) ? (diff_temp - 256) : diff_temp;
        int32_t current = prev + diff;
        output[i] = static_cast<int8_t>(current);
        prev = current;
    }
}

/**
 * Reconstruct tile from residual using predictor
 */
__device__ void reconstructDevice(
    int8_t* tile, uint32_t rows, uint32_t cols,
    const int8_t* left, const int8_t* top, PredictorMode mode)
{
    for (uint32_t r = 0; r < rows; r++) {
        for (uint32_t c = 0; c < cols; c++) {
            int idx = r * cols + c;
            int8_t pred = 0;
            
            switch (mode) {
                case PRED_LEFT:
                    pred = (c > 0) ? tile[idx - 1] : (left ? left[r] : 0);
                    break;
                    
                case PRED_TOP:
                    pred = (r > 0) ? tile[idx - cols] : (top ? top[c] : 0);
                    break;
                    
                case PRED_AVG: {
                    int8_t left_val = (c > 0) ? tile[idx - 1] : (left ? left[r] : 0);
                    int8_t top_val = (r > 0) ? tile[idx - cols] : (top ? top[c] : 0);
                    pred = (left_val + top_val) / 2;
                    break;
                }
                
                case PRED_PLANAR: {
                    int8_t left_val = (c > 0) ? tile[idx - 1] : (left ? left[r] : 0);
                    int8_t top_val = (r > 0) ? tile[idx - cols] : (top ? top[c] : 0);
                    int8_t tl_val = 0;
                    if (r > 0 && c > 0) {
                        tl_val = tile[idx - cols - 1];
                    }
                    pred = left_val + top_val - tl_val;
                    break;
                }
            }
            
            // Residual -> Original
            tile[idx] = tile[idx] + pred;
        }
    }
}

/**
 * Main decode kernel - one thread block per tile
 */
__global__ void decodeKernel(
    const uint8_t* compressed,
    const Header* header,
    const TileMetadata* tile_metadata,
    int8_t* output,
    const uint8_t* global_freq_table)  // Add global freq table parameter
{
    // Each block handles one tile
    uint32_t tile_idx = blockIdx.x;
    uint32_t num_tiles = header->num_tiles_row * header->num_tiles_col;
    
    if (tile_idx >= num_tiles) return;
    
    // Calculate tile position
    uint32_t ty = tile_idx / header->num_tiles_col;
    uint32_t tx = tile_idx % header->num_tiles_col;
    
    uint32_t row_start = ty * header->tile_size;
    uint32_t col_start = tx * header->tile_size;
    uint32_t tile_rows = min(header->tile_size, header->output_rows - row_start);
    uint32_t tile_cols = min(header->tile_size, header->output_cols - col_start);
    
    // Shared memory for tile and frequency table
    extern __shared__ int8_t shared_mem[];
    int8_t* tile_data = shared_mem;
    RANSSymbol* symbols = reinterpret_cast<RANSSymbol*>(shared_mem + 256);
    
    // Thread 0 loads global frequency table into shared memory
    if (threadIdx.x == 0) {
        uint32_t cumul = 0;
        for (int i = 0; i < 256; i++) {
            uint16_t freq = (global_freq_table[i*2 + 0] << 0) | 
                           (global_freq_table[i*2 + 1] << 8);
            symbols[i].start = cumul;
            symbols[i].freq = freq;
            cumul += freq;
        }
    }
    __syncthreads();
    
    // Thread 0 does rANS decode
    if (threadIdx.x == 0) {
        const TileMetadata& meta = tile_metadata[tile_idx];
        const uint8_t* tile_stream = compressed + meta.data_offset;
        
        ransDecodeDevice(tile_stream, meta.data_size, symbols, 
                        tile_data, tile_rows * tile_cols);
    }
    __syncthreads();
    
    // Thread 0 does reconstruction
    if (threadIdx.x == 0) {
        const TileMetadata& meta = tile_metadata[tile_idx];
        
        // Context pointers (simplified - no boundary handling for now)
        const int8_t* left = nullptr;
        const int8_t* top = nullptr;
        
        reconstructDevice(tile_data, tile_rows, tile_cols, left, top,
                         static_cast<PredictorMode>(meta.predictor_mode));
    }
    __syncthreads();
    
    // All threads cooperate to write output
    for (uint32_t i = threadIdx.x; i < tile_rows * tile_cols; i += blockDim.x) {
        uint32_t r = i / tile_cols;
        uint32_t c = i % tile_cols;
        output[(row_start + r) * header->output_cols + col_start + c] = tile_data[i];
    }
}

} // namespace codec

/**
 * Host launch function
 */
extern "C" void launchDecodeKernel(
    const uint8_t* d_compressed,
    const codec::Header* d_header,
    const codec::TileMetadata* d_metadata,
    int8_t* d_output,
    hipStream_t stream)
{
    // Copy header to host to get dimensions
    codec::Header h_header;
    hipMemcpy(&h_header, d_header, sizeof(codec::Header), hipMemcpyDeviceToHost);
    
    uint32_t num_tiles = h_header.num_tiles_row * h_header.num_tiles_col;
    size_t metadata_size = num_tiles * sizeof(codec::TileMetadata);
    
    // Global frequency table is after header + metadata
    const uint8_t* d_global_freq_table = d_compressed + sizeof(codec::Header) + metadata_size;
    
    // Shared memory: tile data + frequency table symbols
    size_t shared_mem = h_header.tile_size * h_header.tile_size * sizeof(int8_t) + 
                       256 * sizeof(codec::RANSSymbol);
    
    // Launch one block per tile, 256 threads per block
    codec::decodeKernel<<<num_tiles, 256, shared_mem, stream>>>(
        d_compressed, d_header, d_metadata, d_output, d_global_freq_table
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

